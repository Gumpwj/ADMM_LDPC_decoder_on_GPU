#include "custom_cuda.h"

#define DEBUG 1

bool ERROR_CHECK(hipError_t Status, const char * file, int line)
{
    if(Status != hipSuccess)
    {
        printf("(EE) \n");
        printf("(EE) Error detected in the LDPC decoder (%s : %d)\n", file, line);
        printf("(EE) MSG: %s\n", hipGetErrorString(Status));
        printf("(EE) \n");
        exit( 0 );
        return false;
    }
    return true;
}

void CUDA_MALLOC_HOST(float** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(float);
    Status     = hipHostMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating   Host Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_HOST(double** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(double);
    Status     = hipHostMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating   Host Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_HOST(int** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(int);
    Status     = hipHostMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating   Host Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_HOST(unsigned int** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(unsigned int);
    Status     = hipHostMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating   Host Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_HOST(char** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(char);
    Status     = hipHostMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating   Host Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_DEVICE(float** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(float);
    Status     = hipMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating Device Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_DEVICE(double** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(double);
    Status     = hipMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating Device Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_DEVICE(int** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(int);
    Status     = hipMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating Device Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_DEVICE(unsigned int** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(unsigned int);
    Status     = hipMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating Device Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}

void CUDA_MALLOC_DEVICE(char** ptr, int nbElements){
    hipError_t Status;
    int nbytes = nbElements * sizeof(char);
    Status     = hipMalloc(ptr, nbytes);
#if DEBUG == 1
	//printf("(II)    + Allocating Device Memory, %d elements (%d bytes) adr [0x%8.8X, 0x%8.8X]\n", nbElements, nbytes, *ptr, *ptr+nbElements-1);
#endif
    ERROR_CHECK(Status, __FILE__, __LINE__);
}
