/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda_fp16.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////

#define SWAP_des(x,y) sort2_swap_des_64b(&d##x, &d##y, &p##x, &p##y)
__device__ void sort2_swap_des_64b(double* dx, double* dy, int* px, int* py)
{
	const double Dx = *dx, Dy = (*dy);
	const int   Px = *px, Py = (*py);
	const bool test = (Dx > Dy);
	(*dx) = fmaxf(Dx,Dy);
	(*dy) = fminf(Dx,Dy);
	(*px) = test ? Px : Py;
	(*py) = test ? Py : Px;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

#define SWAP_asc(x,y) sort2_swap_asc_64b(&d##x, &d##y, &p##x, &p##y)
__device__ void sort2_swap_asc_64b(double* dx, double* dy, int* px, int* py)
{
	const double Dx = *dx, Dy = (*dy);
	const int   Px = *px, Py = (*py);
	const bool test = (Dx < Dy);
	(*dx) = fminf(Dx,Dy);
	(*dy) = fmaxf(Dx,Dy);
	(*px) = test ? Px : Py;
	(*py) = test ? Py : Px;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_swap_64b(double d[6], int p[6])
{
	#define SWAP SWAP_des
    int p0 = 0;    int p1 = 1;    int p2 = 2;
    int p3 = 3;    int p4 = 4;    int p5 = 5;
    double d0 = d[0]; double d1 = d[1]; double d2 = d[2];
    double d3 = d[3]; double d4 = d[4]; double d5 = d[5];
    SWAP(1, 2); SWAP(0, 2); SWAP(0, 1); SWAP(4, 5);
    SWAP(3, 5); SWAP(3, 4); SWAP(0, 3); SWAP(1, 4);
    SWAP(2, 5); SWAP(2, 4); SWAP(1, 3); SWAP(2, 3);
    d[0] = d0; d[1] = d1; d[2] = d2;
    d[3] = d3; d[4] = d4; d[5] = d5;
    p[0] = p0; p[1] = p1; p[2] = p2;
    p[3] = p3; p[4] = p4; p[5] = p5;
	#undef SWAP
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_swap_64b(double illr[6], double rllr[6], int ipos[6], int rpos[6])
{
	#define SWAP SWAP_asc
	int  p0 = ipos[0]; int  p1 = ipos[1]; int  p2 = ipos[2];
	int  p3 = ipos[3]; int  p4 = ipos[4]; int  p5 = ipos[5];
    double d0 = illr[0]; double d1 = illr[1]; double d2 = illr[2];
    double d3 = illr[3]; double d4 = illr[4]; double d5 = illr[5];
    SWAP(1, 2); SWAP(0, 2); SWAP(0, 1); SWAP(4, 5);
    SWAP(3, 5); SWAP(3, 4); SWAP(0, 3); SWAP(1, 4);
    SWAP(2, 5); SWAP(2, 4); SWAP(1, 3); SWAP(2, 3);
    rllr[0] = d0; rllr[1] = d1; rllr[2] = d2;
    rllr[3] = d3; rllr[4] = d4; rllr[5] = d5;
    rpos[0] = p0; rpos[1] = p1; rpos[2] = p2;
    rpos[3] = p3; rpos[4] = p4; rpos[5] = p5;
	#undef SWAP
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_rank_order_reg_64b(double llr[ ], int pos[ ])
{
	const double x0 = llr[0]; const double x1 = llr[1]; const double x2 = llr[2];
    const double x3 = llr[3]; const double x4 = llr[4]; const double x5 = llr[5];
    const int   o0 = (x0< x1) + (x0< x2) + (x0< x3) + (x0< x4) + (x0<x5);
    const int   o1 = (x1<=x0) + (x1< x2) + (x1< x3) + (x1< x4) + (x1<x5);
    const int   o2 = (x2<=x0) + (x2<=x1) + (x2< x3) + (x2< x4) + (x2<x5);
    const int   o3 = (x3<=x0) + (x3<=x1) + (x3<=x2) + (x3< x4) + (x3<x5);
    const int   o4 = (x4<=x0) + (x4<=x1) + (x4<=x2) + (x4<=x3) + (x4<x5);
    const int   o5 = 15 - (o0 + o1 + o2 + o3 + o4);
    llr[o0]=x0; llr[o1]=x1; llr[o2]=x2; llr[o3]=x3; llr[o4]=x4; llr[o5]=x5;
    pos[o0]= 0; pos[o1]= 1; pos[o2]= 2; pos[o3]= 3; pos[o4]= 4; pos[o5]= 5;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_rank_order_reg_modif_64b(double illr[ ], double rllr[ ], int ipos[ ], int rpos[ ])
{
	const double x0 = illr[0], x1 = illr[1], x2 = illr[2];
	const double x3 = illr[3], x4 = illr[4], x5 = illr[5];
	const int   o0 = (x0> x1) + (x0> x2) + (x0> x3) + (x0> x4) + (x0>x5);
	const int   o1 = (x1>=x0) + (x1> x2) + (x1> x3) + (x1> x4) + (x1>x5);
	const int   o2 = (x2>=x0) + (x2>=x1) + (x2> x3) + (x2> x4) + (x2>x5);
	const int   o3 = (x3>=x0) + (x3>=x1) + (x3>=x2) + (x3> x4) + (x3>x5);
	const int   o4 = (x4>=x0) + (x4>=x1) + (x4>=x2) + (x4>=x3) + (x4>x5);
	const int   o5 = 15 - (o0 + o1 + o2 + o3 + o4);
	rllr[o0]=x0;      rllr[o1]=x1;      rllr[o2]=x2;      rllr[o3]=x3;      rllr[o4]=x4;      rllr[o5]=x5;
	rpos[o0]=ipos[0]; rpos[o1]=ipos[1]; rpos[o2]=ipos[2]; rpos[o3]=ipos[3]; rpos[o4]=ipos[4]; rpos[o5]=ipos[5];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__shared__ int sdata[128*12]; // > 512

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void projection_deg6_64b(double llr[], double results[])
{
	const int length = 6;
	bool finished    = false;

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

	int AllZero = (llr[0] <= 0);
	int AllOne  = (llr[0] <= 0);

	#pragma unroll
	for(int i = 1; i < length; i++)
	{
		AllZero = AllZero + (llr[i] <= 0);
		AllOne  = AllOne  + (llr[i] <= 0);
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

	finished = (AllZero == length);

    __syncthreads( );

    /////////////////////////////////////////////////////////////////////////////////////////////////////////

    bool test = (finished == false) && (AllOne == length) && ((length&0x01) == 0);
	#pragma unroll
	for(int i = 0; i < length; i++)
		results[i] = (test == true) ? 1.0f : 0.0f;
	finished = finished | test;

    __syncthreads( );
//Twirl
	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    double constituent = 0;
	double llrClip[6];
	int   zSorti[6] = {0, 1, 2, 3, 4, 5};

	sort6_swap_64b(llr, zSorti);

	#pragma unroll
	for(int i = 0; i < length; i++)// project on the [0,1]^d cube
	{
		const double vMax = fminf(fmaxf(llr[i], 0.0f), 1.0f);
		llrClip[i]       = vMax;
		constituent     += vMax;
	}

	int r = (int)constituent;
    r     = r & 0xFFFFFFFE;//- (r & 0x01);

	double sum_Clip = llrClip[0];
	for(int i = 1; i < length; i++)
	{
		sum_Clip += (i <  r+1) ? llrClip[i] : -llrClip[i];
	}

	// affectation conditionnelle des resultats
	bool valid = ( finished == false ) && (sum_Clip <= r);
	#pragma unroll
	for(int i = 0; i < length; i++)
		results[zSorti[i]] = (valid == true) ? llrClip[i] : results[zSorti[i]];
	finished = finished || valid;

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    __syncthreads();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    if( finished == false )
	{
		double beta     = 0;
		double beta_max = (r + 2 <= length) ? (llr[r] - llr[r+1])/2 : llr[r]; // assign beta_max

	    // sorting zBetaRep
		int   zSorti_m[6] = {0, 1, 2, 3, 4, 5};
		double T_in[6];
	    double T_out[6];
	    int   order_out[6];

		#pragma unroll
		for(int i = 0; i < length; i++)
	        T_in[i] = (i < r+1) ? llr[i] - 1.0f : -llr[i];

		sort6_rank_order_reg_modif_64b (T_in, T_out, zSorti_m, order_out);

		int clip_idx  = -1;
		int zero_idx  =  0;
		int idx_start =  0;
		int idx_end   = -1;

		#pragma unroll 6
		for(int i = 0; i < length; i++)
		{
			clip_idx  += (llr[i]   >     1.0f);
			zero_idx  += (llr[i]   >= -1e-10f);
			idx_start += (T_out[i] <   1e-10f);
			idx_end   += (T_out[i] < beta_max);
		}

		double active_sum = 0;

		#pragma unroll 6
		for(int i = 0;i < length; i++)
		{
			active_sum += (i > clip_idx && i <= r      ) ? llr[i] : 0.0f;
			active_sum -= (i > r        && i < zero_idx) ? llr[i] : 0.0f;
		}

		double total_sum           = active_sum + clip_idx + 1;
		int previous_clip_idx     = clip_idx;
		int previous_zero_idx     = zero_idx;
		double previous_active_sum = active_sum;
		bool change_pre           = false;

		for(int i = idx_start; i <= idx_end; i++)// pour tous les beta entre 0 et beta_max
		{
			if(change_pre)
			{
				// save previous things
				previous_clip_idx   = clip_idx;
				previous_zero_idx   = zero_idx;
				previous_active_sum = active_sum;
			}
			change_pre = false;

			beta = T_out[i];
			clip_idx   -= (order_out[i] <= r);
			zero_idx   += (order_out[i] >  r);
			active_sum += (order_out[i] <= r) ? llr[order_out[i]] : -llr[order_out[i]];

			if (i < length - 1)
			{
				if (beta != T_out[i+1])
				{
					total_sum  = (clip_idx + 1) + active_sum - beta * (zero_idx - clip_idx - 1);
					change_pre = true;
					if(total_sum < r)
						break;
				}

			}
			else if (i == length - 1)
			{
				total_sum  = (clip_idx + 1)  + active_sum - beta * (zero_idx - clip_idx - 1);
				change_pre = true;
			}
		}

		clip_idx   = (total_sum > r) ? clip_idx   : previous_clip_idx;
		active_sum = (total_sum > r) ? active_sum : previous_active_sum;
		zero_idx   = (total_sum > r) ? zero_idx   : previous_zero_idx;
		beta       = -(r - clip_idx - 1 - active_sum)/(zero_idx - clip_idx - 1);

		#pragma unroll 6
		for(int i = 0; i < length; i++)
		{
			const double vA = llr[i];
			const double vD = (i <= r) ? vA - beta : vA + beta;
			results[zSorti[i]] = fminf(fmaxf(vD, 0.0f), 1.0f);
		}
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    __syncthreads();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

#define DOUBLE2

__global__ void ADMM_InitArrays_64b(double* LZr, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
    	double2* ptr = reinterpret_cast<double2*>(LZr);
    	ptr[i]      = make_double2(0.00f, 0.50f);
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ADMM_VN_kernel_deg3(
	const double* _LogLikelihoodRatio, double* OutputFromDecoder, double* LZr, const unsigned int *t_row, int N)
{
    const int i             = blockDim.x * blockIdx.x + threadIdx.x;
	const double mu      = 3.0f;
	const double  alpha  = 0.8;
	const double _amu_   = alpha / mu;
	const double _2_amu_ = _amu_+ _amu_;
    const double factor  = 1.0f / (3.0f - _2_amu_);
    const int   degVn       = 3;

    if (i < N){
        double temp                  = -_LogLikelihoodRatio[i]; // <= OK
        const int frame_offset      = (i%2640);
        const int num_trame         = (i/2640);
        const ushort4  off          = reinterpret_cast<ushort4*>((unsigned int *)t_row)[ frame_offset ];
        const unsigned short tab[4] = {off.x, off.y, off.z, off.w};

        #pragma unroll 3
        for(int k = 0; k < degVn; k++)
        {
        	const int pos = 3 * i + k;
        	const int off = tab[k];//t_row[ pos ];
#ifdef DOUBLE2
        	const double2* ptr = reinterpret_cast<double2*>(LZr);
         	const double2 data = ptr[ (8440 * num_trame) + off ];
                temp       += (data.y + data.x);
#else
                temp       += ( zReplica[ off ] + Lambda[ off ] );
#endif
        }
        const double xx       = (temp  -  _amu_) * factor;
        OutputFromDecoder[i] = fmaxf(fminf(xx, 1.0f), 0.0f);
    }
}



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



__global__ void ADMM_CN_kernel_deg6(
	const double *OutputFromDecoder, double *LZr, const unsigned int *t_col1, int *cn_synrome, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x; // NUMERO DU CHECK NODE A CALCULER
	const double rho      = 1.9f;
	const double un_m_rho = 1.0f - rho;
	const int   degCn    = 6;
        double v_proj[6], ztemp [6];

    if (i < N){
        const int frame_offset = i%1320;
        const int trame_start  = 2640 * (i/1320);

    	int syndrom = 0;
        #pragma unroll
        for(int k = 0; k < degCn; k++)
        {
            const int ind      = degCn * i + k;
            const int offset   = t_col1[ degCn * frame_offset + k ];
            const double xpred  = OutputFromDecoder[ trame_start + offset ];
            syndrom           += (xpred > 0.5);
            reinterpret_cast<double*>(sdata)[threadIdx.x + 128 * k] = xpred;
#ifdef DOUBLE2
        	const double2* ptr = reinterpret_cast<double2*>(LZr);
        	const double2 data = ptr[ ind ];
            v_proj[k]         = (rho * xpred) + (un_m_rho * data.y) - data.x;
#else
            v_proj[k]         = (rho * xpred) + (un_m_rho * zReplica[ind]) - Lambda[ind];
#endif
        }
        cn_synrome[i] = syndrom & 0x01;

        projection_deg6_64b(v_proj, ztemp);

        #pragma unroll
        for(int k = 0; k < degCn; k++)
        {
            const int ind     = degCn * i + k;
            const double xpred = reinterpret_cast<double*>(sdata)[threadIdx.x + 128 * k];
#ifdef DOUBLE2
            double2* ptr = reinterpret_cast<double2*>(LZr);
            double2 data = ptr[ ind ];
            double x     = data.x + (rho * (ztemp[k] - xpred) + un_m_rho * (ztemp[k] - data.y));
            ptr[ ind ]  = make_double2(x, ztemp[k]);
#else
            Lambda[ind]    = Lambda[ind] + (rho * (ztemp[k] - xpred) + un_m_rho * (ztemp[k] - zReplica[ind]));
            zReplica[ind]  = ztemp[k];
#endif
        }
    }
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

