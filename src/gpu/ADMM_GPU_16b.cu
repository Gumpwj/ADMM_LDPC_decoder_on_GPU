/*
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_fp16.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////

#define SWAP_des(x,y) sort2_swap_des(&d##x, &d##y, &p##x, &p##y)
__device__ void sort2_swap_des(float* dx, float* dy, int* px, int* py)
{
	const float Dx = *dx, Dy = (*dy);
	const int   Px = *px, Py = (*py);
	const bool test = (Dx > Dy);
	(*dx) = fmaxf(Dx,Dy);
	(*dy) = fminf(Dx,Dy);
	(*px) = test ? Px : Py;
	(*py) = test ? Py : Px;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

#define SWAP_asc(x,y) sort2_swap_asc(&d##x, &d##y, &p##x, &p##y)
__device__ void sort2_swap_asc(float* dx, float* dy, int* px, int* py)
{
	const float Dx = *dx, Dy = (*dy);
	const int   Px = *px, Py = (*py);
	const bool test = (Dx < Dy);
	(*dx) = fminf(Dx,Dy);
	(*dy) = fmaxf(Dx,Dy);
	(*px) = test ? Px : Py;
	(*py) = test ? Py : Px;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_swap(float d[6], int p[6])
{
	#define SWAP SWAP_des
	int p0 = 0;    int p1 = 1;    int p2 = 2;
	int p3 = 3;    int p4 = 4;    int p5 = 5;
    float d0 = d[0]; float d1 = d[1]; float d2 = d[2];
    float d3 = d[3]; float d4 = d[4]; float d5 = d[5];
    SWAP(1, 2); SWAP(0, 2); SWAP(0, 1); SWAP(4, 5);
    SWAP(3, 5); SWAP(3, 4); SWAP(0, 3); SWAP(1, 4);
    SWAP(2, 5); SWAP(2, 4); SWAP(1, 3); SWAP(2, 3);
    d[0] = d0; d[1] = d1; d[2] = d2; d[3] = d3; d[4] = d4; d[5] = d5;
    p[0] = p0; p[1] = p1; p[2] = p2; p[3] = p3; p[4] = p4; p[5] = p5;
	#undef SWAP
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_swap(float illr[6], float rllr[6], int ipos[6], int rpos[6])
{
	#define SWAP SWAP_asc
	int  p0 = ipos[0]; int  p1 = ipos[1]; int  p2 = ipos[2];
	int  p3 = ipos[3]; int  p4 = ipos[4]; int  p5 = ipos[5];
    float d0 = illr[0]; float d1 = illr[1]; float d2 = illr[2];
    float d3 = illr[3]; float d4 = illr[4]; float d5 = illr[5];
    SWAP(1, 2); SWAP(0, 2); SWAP(0, 1); SWAP(4, 5);
    SWAP(3, 5); SWAP(3, 4); SWAP(0, 3); SWAP(1, 4);
    SWAP(2, 5); SWAP(2, 4); SWAP(1, 3); SWAP(2, 3);
    rllr[0] = d0; rllr[1] = d1; rllr[2] = d2; rllr[3] = d3; rllr[4] = d4; rllr[5] = d5;
    rpos[0] = p0; rpos[1] = p1; rpos[2] = p2; rpos[3] = p3; rpos[4] = p4; rpos[5] = p5;
	#undef SWAP
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_rank_order_reg(float llr[ ], int pos[ ])
{
	const float x0 = llr[0]; const float x1 = llr[1]; const float x2 = llr[2];
    const float x3 = llr[3]; const float x4 = llr[4]; const float x5 = llr[5];
    const int   o0 = (x0< x1) + (x0< x2) + (x0< x3) + (x0< x4) + (x0<x5);
    const int   o1 = (x1<=x0) + (x1< x2) + (x1< x3) + (x1< x4) + (x1<x5);
    const int   o2 = (x2<=x0) + (x2<=x1) + (x2< x3) + (x2< x4) + (x2<x5);
    const int   o3 = (x3<=x0) + (x3<=x1) + (x3<=x2) + (x3< x4) + (x3<x5);
    const int   o4 = (x4<=x0) + (x4<=x1) + (x4<=x2) + (x4<=x3) + (x4<x5);
    const int   o5 = 15 - (o0 + o1 + o2 + o3 + o4);
    llr[o0]=x0; llr[o1]=x1; llr[o2]=x2; llr[o3]=x3; llr[o4]=x4; llr[o5]=x5;
    pos[o0]= 0; pos[o1]= 1; pos[o2]= 2; pos[o3]= 3; pos[o4]= 4; pos[o5]= 5;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void sort6_rank_order_reg_modif(float illr[ ], float rllr[ ], int ipos[ ], int rpos[ ])
{
	const float x0 = illr[0], x1 = illr[1], x2 = illr[2];
	const float x3 = illr[3], x4 = illr[4], x5 = illr[5];
	const int   o0 = (x0> x1) + (x0> x2) + (x0> x3) + (x0> x4) + (x0>x5);
	const int   o1 = (x1>=x0) + (x1> x2) + (x1> x3) + (x1> x4) + (x1>x5);
	const int   o2 = (x2>=x0) + (x2>=x1) + (x2> x3) + (x2> x4) + (x2>x5);
	const int   o3 = (x3>=x0) + (x3>=x1) + (x3>=x2) + (x3> x4) + (x3>x5);
	const int   o4 = (x4>=x0) + (x4>=x1) + (x4>=x2) + (x4>=x3) + (x4>x5);
	const int   o5 = 15 - (o0 + o1 + o2 + o3 + o4);
	rllr[o0]=x0;      rllr[o1]=x1;      rllr[o2]=x2;      rllr[o3]=x3;      rllr[o4]=x4;      rllr[o5]=x5;
	rpos[o0]=ipos[0]; rpos[o1]=ipos[1]; rpos[o2]=ipos[2]; rpos[o3]=ipos[3]; rpos[o4]=ipos[4]; rpos[o5]=ipos[5];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////

__shared__ int sdata[128*6]; // > 512

////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ void projection_deg6(float llr[], float results[])
{
	const int length = 6;
	bool finished    = false;

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

	int AllZero = (llr[0] <= 0);
	int AllOne  = (llr[0] <= 0);

	#pragma unroll
	for(int i = 1; i < length; i++)
	{
		AllZero = AllZero + (llr[i] <= 0);
		AllOne  = AllOne  + (llr[i] <= 0);
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

	finished = (AllZero == length);

    __syncthreads( );

    /////////////////////////////////////////////////////////////////////////////////////////////////////////

    bool test = (finished == false) && (AllOne == length) && ((length&0x01) == 0);
	#pragma unroll
	for(int i = 0; i < length; i++)
		results[i] = (test == true) ? 1.0f : 0.0f;
	finished = finished | test;

    __syncthreads( );

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    float constituent = 0;
	float llrClip[6];
	int   zSorti[6] = {0, 1, 2, 3, 4, 5};

	sort6_swap(llr, zSorti);

	#pragma unroll
	for(int i = 0; i < length; i++)// project on the [0,1]^d cube
	{
		const float vMax = fminf(fmaxf(llr[i], 0.0f), 1.0f);
		llrClip[i]       = vMax;
		constituent     += vMax;
	}

	int r = (int)constituent;
    r     = r & 0xFFFFFFFE;//- (r & 0x01);

	float sum_Clip = llrClip[0];
	for(int i = 1; i < length; i++)
	{
		sum_Clip += (i <  r+1) ? llrClip[i] : -llrClip[i];
	}

	// affectation conditionnelle des resultats
	bool valid = ( finished == false ) && (sum_Clip <= r);
	#pragma unroll
	for(int i = 0; i < length; i++)
		results[zSorti[i]] = (valid == true) ? llrClip[i] : results[zSorti[i]];
	finished = finished || valid;

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    __syncthreads();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    if( finished == false )
	{
		float beta     = 0;
		float beta_max = (r + 2 <= length) ? (llr[r] - llr[r+1])/2 : llr[r]; // assign beta_max

	    // sorting zBetaRep
		int   zSorti_m[6] = {0, 1, 2, 3, 4, 5};
		float T_in[6];
	    float T_out[6];
	    int   order_out[6];

		#pragma unroll
		for(int i = 0; i < length; i++)
	        T_in[i] = (i < r+1) ? llr[i] - 1.0f : -llr[i];

		sort6_rank_order_reg_modif (T_in, T_out, zSorti_m, order_out);

		int clip_idx  = -1;
		int zero_idx  =  0;
		int idx_start =  0;
		int idx_end   = -1;

		#pragma unroll 6
		for(int i = 0; i < length; i++)
		{
			clip_idx  += (llr[i]   >     1.0f);
			zero_idx  += (llr[i]   >= -1e-10f);
			idx_start += (T_out[i] <   1e-10f);
			idx_end   += (T_out[i] < beta_max);
		}

		float active_sum = 0;

		#pragma unroll 6
		for(int i = 0;i < length; i++)
		{
			active_sum += (i > clip_idx && i <= r      ) ? llr[i] : 0.0f;
			active_sum -= (i > r        && i < zero_idx) ? llr[i] : 0.0f;
		}

		float total_sum           = active_sum + clip_idx + 1;
		int previous_clip_idx     = clip_idx;
		int previous_zero_idx     = zero_idx;
		float previous_active_sum = active_sum;
		bool change_pre           = false;

		for(int i = idx_start; i <= idx_end; i++)// pour tous les beta entre 0 et beta_max
		{
			if(change_pre)
			{
				// save previous things
				previous_clip_idx   = clip_idx;
				previous_zero_idx   = zero_idx;
				previous_active_sum = active_sum;
			}
			change_pre = false;

			beta = T_out[i];
			clip_idx   -= (order_out[i] <= r);
			zero_idx   += (order_out[i] >  r);
			active_sum += (order_out[i] <= r) ? llr[order_out[i]] : -llr[order_out[i]];

			if (i < length - 1)
			{
				if (beta != T_out[i+1])
				{
					total_sum  = (clip_idx + 1) + active_sum - beta * (zero_idx - clip_idx - 1);
					change_pre = true;
					if(total_sum < r)
						break;
				}

			}
			else if (i == length - 1)
			{
				total_sum  = (clip_idx + 1)  + active_sum - beta * (zero_idx - clip_idx - 1);
				change_pre = true;
			}
		}

		clip_idx   = (total_sum > r) ? clip_idx   : previous_clip_idx;
		active_sum = (total_sum > r) ? active_sum : previous_active_sum;
		zero_idx   = (total_sum > r) ? zero_idx   : previous_zero_idx;
		beta       = -(r - clip_idx - 1 - active_sum)/(zero_idx - clip_idx - 1);

		#pragma unroll 6
		for(int i = 0; i < length; i++)
		{
			const float vA = llr[i];
			const float vD = (i <= r) ? vA - beta : vA + beta;
			results[zSorti[i]] = fminf(fmaxf(vD, 0.0f), 1.0f);
		}
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////

    __syncthreads();

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ADMM_InitArrays_16b(float* LZr, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
    	__half   t1  = __float2half  ( 0.00f  ); // Lambda
    	__half   t2  = __float2half  ( 0.50f  ); // zReplica
    	__half2* ptr = reinterpret_cast<__half2*>(LZr);
    	ptr[i]       = __halves2half2( t1, t2 );
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ADMM_VN_kernel_deg3_16b(
	const float* _LogLikelihoodRatio, float* OutputFromDecoder, float* LZr, const unsigned int *t_row, int N)
{
    const int i             = blockDim.x * blockIdx.x + threadIdx.x;
	const float mu      = 3.0f;
	const float  alpha  = 0.8;
	const float _amu_   = alpha / mu;
	const float _2_amu_ = _amu_+ _amu_;
    const float factor  = 1.0f / (3.0f - _2_amu_);
    const int   degVn       = 3;
	const __half2* ptr      = reinterpret_cast<__half2*>(LZr);

    if (i < N){
        float temp                  = -_LogLikelihoodRatio[i];
        const int frame_offset      = (i%2640);
        const int num_trame         = (i/2640);
        const ushort4  off          = reinterpret_cast<ushort4*>((unsigned int *)t_row)[ frame_offset ];
        const unsigned short tab[4] = {off.x, off.y, off.z, off.w};

        #pragma unroll 3
        for(int k = 0; k < degVn; k++)
        {
        	const int off = tab[k];
        	const __half2 data = ptr[ (7920 * num_trame) + off ];
            temp              += ( __high2float(data) + __low2float(data) );
        }
        const float xx       = (temp  -  _amu_) * factor;
        OutputFromDecoder[i] = fmaxf(fminf(xx, 1.0f), 0.0f);
    }
}

__global__ void ADMM_CN_kernel_deg6_16b(
	const float *OutputFromDecoder, float *Lzr, const unsigned int *t_col1, int *cn_synrome, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x; // NUMERO DU CHECK NODE A CALCULER
    const float rho      = 1.9f;
    const float un_m_rho = 1.0f - rho;
    const int   degCn    = 6;
    float v_proj[6];
    float ztemp [6];
    __half2* ptr = reinterpret_cast<__half2*>(Lzr);
    float*   PTR = reinterpret_cast<float*>(sdata);

    if (i < N){
        const int frame_offset = i%1320;
        const int trame_start  = 2640 * (i/1320);

    	int syndrom = 0;
#if 1
    	unsigned short* cptr = (unsigned short*)t_col1;//)[]);
        const uint3 offset   = reinterpret_cast<uint3*>( cptr )[ frame_offset ];
        const unsigned int    TAB[3] = {offset.x, offset.y, offset.z};
        const unsigned short* tab    = (const unsigned short*)TAB;
#endif
    	#pragma unroll 6
        for(int k = 0; k < degCn; k++)
        {
            const int ind      = degCn * i + k;
#if 1
            const float xpred  = OutputFromDecoder[ trame_start + tab[ k ] ];
#else
            const int offset   = t_col1[ degCn * frame_offset + k ];
            const float xpred  = OutputFromDecoder[ trame_start + offset ];
#endif
            syndrom           += (xpred > 0.5);

            PTR[threadIdx.x + 128 * k] = xpred;
        	const __half2 data         = ptr[ ind ];
        	v_proj[k]                  = (rho * xpred) + (un_m_rho * __high2float(data)) - __low2float(data);
        }
        cn_synrome[i] = syndrom & 0x01;

        projection_deg6(v_proj, ztemp);

        #pragma unroll 6
        for(int k = 0; k < degCn; k++)
        {
            const int ind      = degCn * i + k;
            const float  xpred = PTR[threadIdx.x + 128 * k];
            const __half2 data = ptr[ ind ];
            float x            = __low2float(data) + (rho * (ztemp[k] - xpred) + un_m_rho * (ztemp[k] - __high2float(data)));
            ptr[ ind ]         = __halves2half2( __float2half(x), __float2half(ztemp[k]) );
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ADMM_VN_kernel_deg3_16b_mod(
	const float* _LogLikelihoodRatio, float* OutputFromDecoder, float* LZr, const unsigned int *t_row, int N)
{
    const int i             = blockDim.x * blockIdx.x + threadIdx.x;
	const float mu      = 3.0f;
	const float  alpha  = 0.8;
	const float _amu_   = alpha / mu;
	const float _2_amu_ = _amu_+ _amu_;
    const float factor  = 1.0f / (3.0f - _2_amu_);
    const int   degVn       = 3;
	const __half2* ptr      = reinterpret_cast<__half2*>(LZr);

    if (i < N){
        float temp                  = -_LogLikelihoodRatio[i];
        const int frame_offset      = (i%2640);
        const int num_trame         = (i/2640);
        const ushort4  off          = reinterpret_cast<ushort4*>((unsigned int *)t_row)[ frame_offset ];
        const unsigned short tab[4] = {off.x, off.y, off.z, off.w};

        #pragma unroll 3
        for(int k = 0; k < degVn; k++)
        {
        	const int off = tab[k];
        	const __half2 data = ptr[ (8448 * num_trame) + off ];
            temp              += ( __high2float(data) + __low2float(data) );
        }
        const float xx       = (temp  -  _amu_) * factor;
        OutputFromDecoder[i] = fmaxf(fminf(xx, 1.0f), 0.0f);
    }
}


__global__ void ADMM_CN_kernel_deg6_16b_mod(
	const float *OutputFromDecoder, float *Lzr, const unsigned int *t_col1, int *cn_synrome, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x; // NUMERO DU CHECK NODE A CALCULER
    const float rho      = 1.9f;
    const float un_m_rho = 1.0f - rho;
    const int   degCn    = 6;
    float v_proj[6];
    float ztemp [6];
    __half2* ptr = reinterpret_cast<__half2*>(Lzr);
    float*   PTR = reinterpret_cast<float*>(sdata);

    if (i < N){
        const int frame_id     = i/1320;
        const int frame_offset = i%1320;
        const int trame_start  = 2640 * (i/1320);
        const int IND          = 8448 * frame_id; // offset to access mesages from current frame
        const int indice       = IND + 768 * (frame_offset/128) + frame_offset%128;

    	int syndrom = 0;

    	unsigned short* cptr         = (unsigned short*)t_col1;//)[]);
        const uint3 offset           = reinterpret_cast<uint3*>( cptr )[ frame_offset ];
        const unsigned int    TAB[3] = {offset.x, offset.y, offset.z};
        const unsigned short* tab    = (const unsigned short*)TAB;

    	#pragma unroll 6
        for(int k = 0; k < degCn; k++)
        {
            const float xpred          = OutputFromDecoder[ trame_start + tab[ k ] ];
            syndrom                   += (xpred > 0.5);
        	const __half2 data         = ptr[ indice +128 * k ];
        	const auto contribution    = (rho * xpred) + (un_m_rho * __high2float(data)) - __low2float(data);
            v_proj[k]                  = contribution;
            PTR[threadIdx.x + 128 * k] = contribution;

        }
        cn_synrome[i] = syndrom & 0x01;

        projection_deg6(v_proj, ztemp);

        #pragma unroll 6
        for(int k = 0; k < degCn; k++)
        {
            const float  contr      = PTR[threadIdx.x + 128 * k];
            float x                 = ztemp[k] - contr;
            ptr[ indice +128 * k ]  = __halves2half2( __float2half(x), __float2half(ztemp[k]) );
        }
    }
}
